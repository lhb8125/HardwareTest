#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <chrono>
#include <nvml.h>
#include <cstdio>
#include <cstdlib>
#include <string>

static int deviceID = 0;

FILE* test_result;
std::string file_name ("results_" + std::to_string(deviceID) + ".txt");

int gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) 
{
    nvmlReturn_t result;
    unsigned int temp = 0;
    unsigned int clock_freq = 0;

    result = nvmlInit();
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
        printf("Press ENTER to continue...\n");
        getchar();
        return 1;
    }

    nvmlDevice_t device;

    result = nvmlDeviceGetHandleByIndex(deviceID, &device);
    if (NVML_SUCCESS != result)
    { 
        printf("Failed to get handle for device %i: %s\n", deviceID, nvmlErrorString(result));
        result = nvmlShutdown();
        if (NVML_SUCCESS != result)
            printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));
        printf("Press ENTER to continue...\n");
        getchar();
        return 1;
    }

    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
    const double G_operations = 2*double(m)/1000*double(k)/1000*double(n)/1000;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    test_result = fopen(file_name.c_str(), "a");
    fprintf(test_result,"Temperature\t TFLOPS\t CLOCKS\t \n");
    fclose(test_result);

    while(temp < 95)
    {   
        hipEventRecord(start);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);    
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        result = nvmlDeviceGetTemperature(device, NVML_TEMPERATURE_GPU, &temp);
        if (NVML_SUCCESS != result) {
            printf("Failed to get temperature of device %i: %s\n", 0, nvmlErrorString(result));
        }

        result = nvmlDeviceGetClockInfo(device, NVML_CLOCK_SM , &clock_freq);
        if (NVML_SUCCESS != result) {
            printf("Failed to get clock frequency of device %i: %s\n", 0, nvmlErrorString(result));
        }
        std::cout << "Temperature : " << temp << "\n";
        std::cout << "TFLOPS : " << G_operations/milliseconds << "\n";          
        std::cout << "CLOCKS : " << clock_freq << "\n";

        test_result = fopen(file_name.c_str(),"a");
        fprintf(test_result,"%d\t %f\t %d\t \n", temp, G_operations/milliseconds, clock_freq);
        fclose(test_result);
    }

    hipblasDestroy(handle);

    result = nvmlShutdown();
    if (NVML_SUCCESS != result)
        printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));
    return 0;
}

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) 
{
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

int main(const int argc, const char **argv)
{
    hipSetDevice(deviceID);

    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    // int m = 28*28, n = 8*128, k = 96;
    int m = 16000, n = 16000, k = 16000;

    nr_rows_A = m;
    nr_rows_C = m;
    nr_cols_A = n;
    nr_rows_B = n;
    nr_cols_B = k;
    nr_cols_C = k;

    float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
    float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
    float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

    GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

    hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
        
    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);      

    // cudaMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),cudaMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);  

    free(h_A);
    free(h_B);
    free(h_C); 

    return 0;
}
